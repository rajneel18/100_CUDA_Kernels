#include <stdio.h>
#include <hip/hip_runtime.h>

    // ReLU Kernel
__global__ void reluKernel(float *d_in, float *d_out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_out[idx] = fmaxf(0.0f, d_in[idx]);
    }
}

int main() {
    int n = 1000;  
    size_t size = n * sizeof(float);
    
    float *h_in = (float*)malloc(size);
    float *h_out = (float*)malloc(size);

    for (int i = 0; i < n; i++) {
        h_in[i] = (i % 2 == 0) ? -i : i;
    }

    float *d_in, *d_out;
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    reluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, n);
    
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("ReLU(%f) = %f\n", h_in[i], h_out[i]);
    }

    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}