#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 4        // Matrix size
#define BLOCK_SIZE 256  // CUDA block size

// CUDA error-checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if(err != hipSuccess) { \
            printf("CUDA error in %s@%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Kernel: Matrix-vector multiplication y = A * x
__global__ void matVecMul(const float *A, const float *x, float *y, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n) {
        float sum = 0.0f;
        for (int j = 0; j < n; j++) {
            sum += A[row * n + j] * x[j];
        }
        y[row] = sum;
    }
}

// Kernel: Vector addition y = y + alpha * x
__global__ void vecAdd(float *y, const float *x, float alpha, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] += alpha * x[i];
    }
}

// Kernel: Vector subtraction y = y - alpha * x
__global__ void vecSub(float *y, const float *x, float alpha, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] -= alpha * x[i];
    }
}

// Kernel: Scale vector x = beta * x
__global__ void vecScale(float *x, float beta, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        x[i] *= beta;
    }
}

// Kernel: Compute dot product using shared memory reduction
__global__ void dotProduct(const float *a, const float *b, float *result, int n) {
    __shared__ float cache[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = 0.0f;

    while (i < n) {
        temp += a[i] * b[i];
        i += blockDim.x * gridDim.x;
    }
    cache[tid] = temp;
    __syncthreads();

    // Reduction in shared memory
    int blockSize = blockDim.x;
    while (blockSize > 1) {
        int half = blockSize / 2;
        if (tid < half) {
            cache[tid] += cache[tid + half];
        }
        __syncthreads();
        blockSize = half;
    }

    if (tid == 0) {
        atomicAdd(result, cache[0]);
    }
}

int main() {
    const int n = N;
    const int matrixSize = n * n * sizeof(float);
    const int vectorSize = n * sizeof(float);

    // Host data: Define a symmetric positive-definite matrix A and vector b.
    float h_A[N * N] = {
         4, 1, 0, 0,
         1, 3, 1, 0,
         0, 1, 2, 1,
         0, 0, 1, 1
    };
    float h_b[N] = {15, 10, 10, 10};
    float h_x[N] = {0}; // Initial guess x = 0

    // Device allocations
    float *d_A, *d_x, *d_b, *d_r, *d_p, *d_Ap;
    CUDA_CHECK(hipMalloc(&d_A, matrixSize));
    CUDA_CHECK(hipMalloc(&d_x, vectorSize));
    CUDA_CHECK(hipMalloc(&d_b, vectorSize));
    CUDA_CHECK(hipMalloc(&d_r, vectorSize));
    CUDA_CHECK(hipMalloc(&d_p, vectorSize));
    CUDA_CHECK(hipMalloc(&d_Ap, vectorSize));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, vectorSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, vectorSize, hipMemcpyHostToDevice));

    // Initialize: r = b - A*x (x=0 => r=b), p = r
    CUDA_CHECK(hipMemcpy(d_r, d_b, vectorSize, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_p, d_r, vectorSize, hipMemcpyDeviceToDevice));

    // Allocate device memory for dot product result
    float *d_dot;
    CUDA_CHECK(hipMalloc(&d_dot, sizeof(float)));

    float rdotr = 0.0f, new_rdotr = 0.0f;

    // Compute initial dot product: rdotr = r^T r
    CUDA_CHECK(hipMemset(d_dot, 0, sizeof(float)));
    dotProduct<<<1, BLOCK_SIZE>>>(d_r, d_r, d_dot, n);
    CUDA_CHECK(hipMemcpy(&rdotr, d_dot, sizeof(float), hipMemcpyDeviceToHost));

    int max_iter = 1000;
    float tol = 1e-6f;
    int k = 0;

    while (sqrt(rdotr) > tol && k < max_iter) {
        // Ap = A * p
        matVecMul<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_p, d_Ap, n);

        // Compute p^T * A * p
        CUDA_CHECK(hipMemset(d_dot, 0, sizeof(float)));
        dotProduct<<<1, BLOCK_SIZE>>>(d_p, d_Ap, d_dot, n);
        float pAp = 0.0f;
        CUDA_CHECK(hipMemcpy(&pAp, d_dot, sizeof(float), hipMemcpyDeviceToHost));

        float alpha = rdotr / pAp;

        // x = x + alpha * p
        vecAdd<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_x, d_p, alpha, n);

        // r = r - alpha * Ap
        vecSub<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_r, d_Ap, alpha, n);

        // Compute new dot product: new_rdotr = r^T * r
        CUDA_CHECK(hipMemset(d_dot, 0, sizeof(float)));
        dotProduct<<<1, BLOCK_SIZE>>>(d_r, d_r, d_dot, n);
        CUDA_CHECK(hipMemcpy(&new_rdotr, d_dot, sizeof(float), hipMemcpyDeviceToHost));

        if (sqrt(new_rdotr) < tol) {
            break;
        }

        float beta = new_rdotr / rdotr;

        // p = r + beta * p
        vecScale<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_p, beta, n);
        vecAdd<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_p, d_r, 1.0f, n);

        rdotr = new_rdotr;
        k++;
    }

    // Copy the solution back to host
    CUDA_CHECK(hipMemcpy(h_x, d_x, vectorSize, hipMemcpyDeviceToHost));
    printf("Conjugate Gradient converged in %d iterations.\n", k);
    printf("Solution x:\n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", h_x[i]);
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_r));
    CUDA_CHECK(hipFree(d_p));
    CUDA_CHECK(hipFree(d_Ap));
    CUDA_CHECK(hipFree(d_dot));

    return 0;
}
