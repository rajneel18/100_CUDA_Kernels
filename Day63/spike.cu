#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_NEURONS 1024
#define TIME_STEPS 100
#define THRESHOLD 1.0f
#define DECAY 0.95f

__global__ void init_random_states(hiprandState *states, int seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < NUM_NEURONS) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void spiking_neuron_sim(float *membrane_potential, int *spike_counts, hiprandState *states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < NUM_NEURONS) {
        hiprandState localState = states[idx];
        for (int t = 0; t < TIME_STEPS; t++) {
            float input_current = hiprand_uniform(&localState) * 0.1f;
            membrane_potential[idx] += input_current;
            if (membrane_potential[idx] >= THRESHOLD) {
                spike_counts[idx]++;
                membrane_potential[idx] = 0.0f;  // Reset after spike
            } else {
                membrane_potential[idx] *= DECAY;
            }
        }
        states[idx] = localState;
    }
}

int main() {
    float *d_membrane_potential;
    int *d_spike_counts;
    hiprandState *d_states;
    
    hipMalloc(&d_membrane_potential, NUM_NEURONS * sizeof(float));
    hipMalloc(&d_spike_counts, NUM_NEURONS * sizeof(int));
    hipMalloc(&d_states, NUM_NEURONS * sizeof(hiprandState));
    
    hipMemset(d_membrane_potential, 0, NUM_NEURONS * sizeof(float));
    hipMemset(d_spike_counts, 0, NUM_NEURONS * sizeof(int));
    
    dim3 blockSize(256);
    dim3 gridSize((NUM_NEURONS + blockSize.x - 1) / blockSize.x);
    
    init_random_states<<<gridSize, blockSize>>>(d_states, time(NULL));
    spiking_neuron_sim<<<gridSize, blockSize>>>(d_membrane_potential, d_spike_counts, d_states);
    
    int *h_spike_counts = (int*)malloc(NUM_NEURONS * sizeof(int));
    hipMemcpy(h_spike_counts, d_spike_counts, NUM_NEURONS * sizeof(int), hipMemcpyDeviceToHost);
    
    int total_spikes = 0;
    for (int i = 0; i < NUM_NEURONS; i++) {
        total_spikes += h_spike_counts[i];
    }
    printf("Total spikes in the network: %d\n", total_spikes);
    
    free(h_spike_counts);
    hipFree(d_membrane_potential);
    hipFree(d_spike_counts);
    hipFree(d_states);
    
    return 0;
}
