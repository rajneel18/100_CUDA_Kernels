#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 500000     
#define NSTEP 1000   
#define NKERNEL 20   

// CUDA Kernel
__global__ void shortKernel(float *out_d, float *in_d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) out_d[idx] = 1.23f * in_d[idx];
}

// Error checking macro
#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (at %s:%d)\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

int main() {
    float *in_d, *out_d;
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t instance;

    
    CHECK_CUDA(hipMalloc(&in_d, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&out_d, N * sizeof(float)));


    CHECK_CUDA(hipStreamCreate(&stream));

    int threads, minGridSize;
    CHECK_CUDA(hipOccupancyMaxPotentialBlockSize(&minGridSize, &threads, shortKernel, 0, N));
    int blocks = (N + threads - 1) / threads;

    CHECK_CUDA(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    for (int ikrnl = 0; ikrnl < NKERNEL; ikrnl++) {
        shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
    }

    // create graph
    CHECK_CUDA(hipStreamEndCapture(stream, &graph));
    CHECK_CUDA(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));

    // Warm-up Execution
    CHECK_CUDA(hipGraphLaunch(instance, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

    hipEvent_t start, stop;
    float elapsedTime;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, stream));

    for (int istep = 0; istep < NSTEP; istep++) {
        CHECK_CUDA(hipGraphLaunch(instance, stream));
    }

    CHECK_CUDA(hipEventRecord(stop, stream));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&elapsedTime, start, stop));

    float timePerKernel = (elapsedTime * 1000) / (NSTEP * NKERNEL);
    printf("Time taken per kernel using CUDA Graphs: %.3f microseconds\n", timePerKernel);

    CHECK_CUDA(hipGraphDestroy(graph));
    CHECK_CUDA(hipGraphExecDestroy(instance));
    CHECK_CUDA(hipFree(in_d));
    CHECK_CUDA(hipFree(out_d));
    CHECK_CUDA(hipStreamDestroy(stream));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return 0;
}
