#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 500000    
#define NSTEP 1000  
#define NKERNEL 20  

#define THREADS_PER_BLOCK 512

__global__ void shortKernel(float *out_d, float *in_d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out_d[idx] = 1.23f * in_d[idx];
    }
}

void checkCuda(hipError_t result, const char *msg) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s (%s)\n", msg, hipGetErrorString(result));
        exit(-1);
    }
}

int main() {
    float *d_in, *d_out;
    hipStream_t stream;
    
    checkCuda(hipMalloc(&d_in, N * sizeof(float)), "Alloc d_in");
    checkCuda(hipMalloc(&d_out, N * sizeof(float)), "Alloc d_out");

    checkCuda(hipStreamCreate(&stream), "Stream creation");

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start), "Event start creation");
    checkCuda(hipEventCreate(&stop), "Event stop creation");
    checkCuda(hipEventRecord(start), "Start event recording");

    for (int istep = 0; istep < NSTEP; istep++) {
        for (int ikrnl = 0; ikrnl < NKERNEL; ikrnl++) {
            shortKernel<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(d_out, d_in);
            hipStreamSynchronize(stream);
        }
    }

    checkCuda(hipEventRecord(stop), "Stop event recording");
    checkCuda(hipEventSynchronize(stop), "Stop event synchronization");

    float milliseconds = 0;
    checkCuda(hipEventElapsedTime(&milliseconds, start, stop), "Elapsed time calculation");

    printf("Time taken per kernel: %f microseconds\n", (milliseconds * 1000) / (NSTEP * NKERNEL));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
