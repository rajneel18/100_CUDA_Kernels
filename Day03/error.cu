#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main(){
    int *d_A;
    size_t size = 100000000000 * sizeof(int);

    hipError_t err = hipMalloc((void**)&d_A, size);

    if (err!= hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    else{
        printf("Memory allocated successfully.\n");
    }

    hipFree(d_A);

    return 0;

}