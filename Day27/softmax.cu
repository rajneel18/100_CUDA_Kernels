#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void softmax(float* input, float* output, int N) {
    int idx = threadIdx.x;
    if (idx >= N) return;

    // Compute exponentials
    float exp_val = expf(input[idx]);

    // Compute sum of exponentials using shared memory
    __shared__ float sum_exp;
    if (idx == 0) sum_exp = 0.0f;
    __syncthreads();

    atomicAdd(&sum_exp, exp_val);
    __syncthreads();

    // Compute softmax
    output[idx] = exp_val / sum_exp;
}

void softmaxHost(float* input, float* output, int N) {
    float *d_input, *d_output;
    size_t size = N * sizeof(float);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording time
    hipEventRecord(start);
    softmax<<<1, N>>>(d_input, d_output, N);
    hipEventRecord(stop);

    // Wait for completion
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);

    // Print execution time
    printf("CUDA Softmax Execution Time: %.6f ms\n", milliseconds);
}

int main() {
    int N = 5;
    float h_input[5] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float h_output[5];

    softmaxHost(h_input, h_output, N);

    printf("Softmax Output:\n");
    for (int i = 0; i < N; i++) {
        printf("%f\n", h_output[i]);
    }
    return 0;
}
