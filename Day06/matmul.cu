
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorMatrixMult(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
      float sum = 0.0f;
      for (int j = 0; j < N; j++) {
         sum += A[i * N + j] * B[j];
      }
      C[i] = sum;
    }
}

int main() {
    const int N = 10;
    float *A, *B, *C;

    // initialize the input matrices
    A = (float *)malloc(N * N * sizeof(float));
    B = (float *)malloc(N * sizeof(float));
    C = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = 1.0f;  // Fill A with 1s
        }
        B[i] = 2.0f;  // Fill B with 2s
        C[i] = 0.0f;  // Initialize C with zeros
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    int blocksize = 256;
    int gridsize = (N + blocksize - 1) / blocksize;
    vectorMatrixMult<<<gridsize, blocksize>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();  // Ensure kernel finishes before moving on

    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%.2f ", A[i * N + j]); // Prints each element with 2 decimal precision
        }
        printf("\n"); // Adds a newline after each row
    }

    printf("C:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", C[i]); // Prints each element with 2 decimal precision
    }
    printf("\n");

    printf("B:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", B[i]); // Prints each element with 2 decimal precision
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
}
