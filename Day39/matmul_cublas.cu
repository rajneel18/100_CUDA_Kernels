#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkCublasErrors(hipblasStatus_t err) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d\n", err);
        exit(EXIT_FAILURE);
    }
}

int main() {
    hipblasHandle_t handle;
    checkCublasErrors(hipblasCreate(&handle));

    int M = 3, N = 2, K = 4;
    float *h_A, *h_B, *h_C;

    h_A = (float *)malloc(M * K * sizeof(float));
    h_B = (float *)malloc(K * N * sizeof(float));
    h_C = (float *)malloc(M * N * sizeof(float));

    // Initialize matrices with different values
    for (int i = 0; i < M; i++)
        for (int j = 0; j < K; j++)
            h_A[i * K + j] = (float)(i * K + j);

    for (int i = 0; i < K; i++)
        for (int j = 0; j < N; j++)
            h_B[i * N + j] = (float)(i * N + j + 1);

    float *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc(&d_A, M * K * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_B, K * N * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_C, M * N * sizeof(float)));

    checkCudaErrors(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    const float alpha = 1.0f, beta = 0.0f;
    checkCublasErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  M, N, K, &alpha,
                                  d_A, M, d_B, K,
                                  &beta, d_C, M));

    checkCudaErrors(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    printf("Matrix A:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            printf("%f ", h_A[i * K + j]);
        }
        printf("\n");
    }

    printf("Matrix B:\n");
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_B[i * N + j]);
        }
        printf("\n");
    }

    printf("Matrix C = A * B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_C[i * N + j]);
        }
        printf("\n");
    }

    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    checkCublasErrors(hipblasDestroy(handle));

    return 0;
}
