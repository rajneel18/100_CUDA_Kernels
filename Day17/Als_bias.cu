#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define NUM_USERS 5
#define NUM_ITEMS 4
#define NUM_FACTORS 2  // Low-rank approximation dimension
#define LAMBDA 0.1     // Regularization parameter
#define MAX_ITER 50    // Maximum iterations
#define LEARNING_RATE 0.01  // Learning rate

// Synthetic test matrix (user-item ratings)
float R[NUM_USERS][NUM_ITEMS] = {
    {5, 3, 0, 1},
    {4, 0, 0, 1},
    {1, 1, 0, 5},
    {1, 0, 0, 4},
    {0, 1, 5, 4}
};

// CUDA kernel to update user factors (P) and user biases (bu)
__global__ void update_user_factors(float *R, float *P, float *Q, float *bu, float *bi, int num_users, int num_items, int num_factors, float lambda, float alpha) {
    int user = blockIdx.x * blockDim.x + threadIdx.x;
    if (user >= num_users) return;

    for (int i = 0; i < num_items; ++i) {
        if (R[user * num_items + i] > 0) {  // Only update for known ratings
            float prediction = bu[user] + bi[i];
            for (int k = 0; k < num_factors; ++k) {
                prediction += P[user * num_factors + k] * Q[k * num_items + i];
            }
            float error = R[user * num_items + i] - prediction;

            // Update user factors and bias
            for (int k = 0; k < num_factors; ++k) {
                P[user * num_factors + k] += alpha * (error * Q[k * num_items + i] - lambda * P[user * num_factors + k]);
            }
            bu[user] += alpha * (error - lambda * bu[user]);
        }
    }
}

// CUDA kernel to update item factors (Q) and item biases (bi)
__global__ void update_item_factors(float *R, float *P, float *Q, float *bu, float *bi, int num_users, int num_items, int num_factors, float lambda, float alpha) {
    int item = blockIdx.x * blockDim.x + threadIdx.x;
    if (item >= num_items) return;

    for (int u = 0; u < num_users; ++u) {
        if (R[u * num_items + item] > 0) {  // Only update for known ratings
            float prediction = bu[u] + bi[item];
            for (int k = 0; k < num_factors; ++k) {
                prediction += P[u * num_factors + k] * Q[k * num_items + item];
            }
            float error = R[u * num_items + item] - prediction;

            // Update item factors and bias
            for (int k = 0; k < num_factors; ++k) {
                Q[k * num_items + item] += alpha * (error * P[u * num_factors + k] - lambda * Q[k * num_items + item]);
            }
            bi[item] += alpha * (error - lambda * bi[item]);
        }
    }
}

// Function to initialize matrices with small random values
void initialize_matrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = 0.1f * ((float)rand() / RAND_MAX);
    }
}

// Host function to run ALS
void runALS(float *R, int num_users, int num_items, int num_factors) {
    float *P, *Q, *bu, *bi, *d_R;
    int size_R = num_users * num_items * sizeof(float);
    int size_P = num_users * num_factors * sizeof(float);
    int size_Q = num_factors * num_items * sizeof(float);
    int size_bu = num_users * sizeof(float);
    int size_bi = num_items * sizeof(float);

    // Allocate host memory
    float *h_P = (float *)malloc(size_P);
    float *h_Q = (float *)malloc(size_Q);
    float *h_bu = (float *)malloc(size_bu);
    float *h_bi = (float *)malloc(size_bi);

    // Initialize P, Q, bu, and bi with random values
    initialize_matrix(h_P, num_users, num_factors);
    initialize_matrix(h_Q, num_factors, num_items);
    for (int i = 0; i < num_users; ++i) h_bu[i] = 0.0f;
    for (int i = 0; i < num_items; ++i) h_bi[i] = 0.0f;

    // Allocate device memory
    hipMalloc((void **)&d_R, size_R);
    hipMalloc((void **)&P, size_P);
    hipMalloc((void **)&Q, size_Q);
    hipMalloc((void **)&bu, size_bu);
    hipMalloc((void **)&bi, size_bi);

    // Copy data to device
    hipMemcpy(d_R, R, size_R, hipMemcpyHostToDevice);
    hipMemcpy(P, h_P, size_P, hipMemcpyHostToDevice);
    hipMemcpy(Q, h_Q, size_Q, hipMemcpyHostToDevice);
    hipMemcpy(bu, h_bu, size_bu, hipMemcpyHostToDevice);
    hipMemcpy(bi, h_bi, size_bi, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSizeUsers = (num_users + blockSize - 1) / blockSize;
    int gridSizeItems = (num_items + blockSize - 1) / blockSize;

    for (int iter = 0; iter < MAX_ITER; ++iter) {
        update_user_factors<<<gridSizeUsers, blockSize>>>(d_R, P, Q, bu, bi, num_users, num_items, num_factors, LAMBDA, LEARNING_RATE);
        hipDeviceSynchronize();
        update_item_factors<<<gridSizeItems, blockSize>>>(d_R, P, Q, bu, bi, num_users, num_items, num_factors, LAMBDA, LEARNING_RATE);
        hipDeviceSynchronize();
    }

    // Copy results back to host
    hipMemcpy(h_P, P, size_P, hipMemcpyDeviceToHost);
    hipMemcpy(h_Q, Q, size_Q, hipMemcpyDeviceToHost);
    hipMemcpy(h_bu, bu, size_bu, hipMemcpyDeviceToHost);
    hipMemcpy(h_bi, bi, size_bi, hipMemcpyDeviceToHost);

    // Print the factor matrices and biases
    printf("Matrix P (User factors):\n");
    for (int i = 0; i < num_users; ++i) {
        for (int j = 0; j < num_factors; ++j) {
            printf("%.4f ", h_P[i * num_factors + j]);
        }
        printf("\n");
    }

    printf("\nMatrix Q (Item factors):\n");
    for (int i = 0; i < num_factors; ++i) {
        for (int j = 0; j < num_items; ++j) {
            printf("%.4f ", h_Q[i * num_items + j]);
        }
        printf("\n");
    }

    printf("\nUser biases:\n");
    for (int i = 0; i < num_users; ++i) {
        printf("%.4f ", h_bu[i]);
    }
    printf("\n\nItem biases:\n");
    for (int i = 0; i < num_items; ++i) {
        printf("%.4f ", h_bi[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_R);
    hipFree(P);
    hipFree(Q);
    hipFree(bu);
    hipFree(bi);

    // Free host memory
    free(h_P);
    free(h_Q);
    free(h_bu);
    free(h_bi);
}

int main() {
    printf("Running ALS with Biases on CUDA with a test matrix...\n");
    runALS(&R[0][0], NUM_USERS, NUM_ITEMS, NUM_FACTORS);
    return 0;
}
