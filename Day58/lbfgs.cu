#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024
#define M 10
#define BLOCK_SIZE 256

__global__ void dotProductKernel(float *a, float *b, float *result, int n) {
    __shared__ float cache[BLOCK_SIZE];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    float temp = 0.0f;
    while (tid < n) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    cache[cacheIndex] = temp;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        atomicAdd(result, cache[0]);
    }
}

__global__ void vectorUpdateKernel(float *x, float *d, float alpha, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        x[tid] += alpha * d[tid];
    }
}

__global__ void computeGradientKernel(float *g, float *Qx, float *b, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        g[tid] = Qx[tid] - b[tid];
    }
}

void lbfgs(float *d_x, float *d_g, float *d_Q, float *d_b, int n, int max_iters) {
    float *s[M], *y[M];
    float alpha[M], rho[M];

    for (int i = 0; i < M; i++) {
        hipMalloc(&s[i], n * sizeof(float));
        hipMalloc(&y[i], n * sizeof(float));
    }

    float *d_Qx, *d_direction, *d_temp, *d_dot_result;
    hipMalloc(&d_Qx, n * sizeof(float));
    hipMalloc(&d_direction, n * sizeof(float));
    hipMalloc(&d_temp, n * sizeof(float));
    hipMalloc(&d_dot_result, sizeof(float));

    dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE);

    for (int iter = 0; iter < max_iters; iter++) {
        hipMemcpy(d_Qx, d_x, n * sizeof(float), hipMemcpyDeviceToDevice);
        computeGradientKernel<<<grid, block>>>(d_g, d_Qx, d_b, n);

        int m = iter < M ? iter : M;
        float beta;
        
        for (int i = m - 1; i >= 0; i--) {
            dotProductKernel<<<grid, block>>>(s[i], d_g, d_dot_result, n);
            hipMemcpy(&alpha[i], d_dot_result, sizeof(float), hipMemcpyDeviceToHost);
            alpha[i] *= rho[i];
        }

        hipMemcpy(d_direction, d_g, n * sizeof(float), hipMemcpyDeviceToDevice);

        for (int i = 0; i < m; i++) {
            dotProductKernel<<<grid, block>>>(y[i], d_direction, d_dot_result, n);
            hipMemcpy(&beta, d_dot_result, sizeof(float), hipMemcpyDeviceToHost);
            beta *= rho[i];

            float scale = alpha[i] - beta;
            vectorUpdateKernel<<<grid, block>>>(d_direction, s[i], scale, n);
        }

        float step_size = 0.01;
        vectorUpdateKernel<<<grid, block>>>(d_x, d_direction, -step_size, n);

        hipMemcpy(s[iter % M], d_x, n * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(y[iter % M], d_g, n * sizeof(float), hipMemcpyDeviceToDevice);
        dotProductKernel<<<grid, block>>>(y[iter % M], s[iter % M], d_dot_result, n);
        hipMemcpy(&rho[iter % M], d_dot_result, sizeof(float), hipMemcpyDeviceToHost);
        rho[iter % M] = 1.0f / rho[iter % M];
    }

    for (int i = 0; i < M; i++) {
        hipFree(s[i]);
        hipFree(y[i]);
    }
    hipFree(d_Qx);
    hipFree(d_direction);
    hipFree(d_temp);
    hipFree(d_dot_result);
}

int main() {
    float *h_x, *h_g, *h_Q, *h_b;
    float *d_x, *d_g, *d_Q, *d_b;

    h_x = (float*)malloc(N * sizeof(float));
    h_g = (float*)malloc(N * sizeof(float));
    h_Q = (float*)malloc(N * sizeof(float));
    h_b = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_x[i] = 0.0f;
        h_b[i] = 1.0f;
        h_Q[i] = 2.0f;
    }

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_g, N * sizeof(float));
    hipMalloc(&d_Q, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));

    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Q, h_Q, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    lbfgs(d_x, d_g, d_Q, d_b, N, 100);

    hipMemcpy(h_x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
    printf("Optimized solution (first 10 values): \n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_x[i]);
    }
    printf("\n");

    free(h_x); free(h_g); free(h_Q); free(h_b);
    hipFree(d_x); hipFree(d_g); hipFree(d_Q); hipFree(d_b);

    return 0;
}
