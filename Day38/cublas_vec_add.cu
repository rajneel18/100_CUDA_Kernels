#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define N 5  
int main() {
    
    float h_x[N] = {1, 2, 3, 4, 5};
    float h_y[N] = {10, 20, 30, 40, 50};
    float alpha = 2.0f; 

    float *d_x, *d_y;
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));

    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // y = alpha * x + y
    hipblasSaxpy(handle, N, &alpha, d_x, 1, d_y, 1);

    hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Resultant y vector: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_y[i]);
    }
    printf("\n");

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
